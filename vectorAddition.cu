// This program demonstrates vector addition using CUDA.

// header files

// std header
#include<stdio.h>

// cuda headers
#include<hip/hip_runtime.h>

// global variables
const int iNumberOfArrayElements = 11444777;

float* hostInput1 = NULL;
float* hostInput2 = NULL;
float* hostOutput = NULL;
float* gold = NULL;

float* deviceInput1 = NULL;
float* deviceInput2 = NULL;
float* deviceOutput = NULL;

// CUDA kernel
__global__ void vectorAdditionGPU(float* in1, float* in2, float* out, int len){

	// code
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i < len){
	
		out[i] = in1[i] + in2[i];
	}

}



// entry-point function.
int main(void){

	// func. declarations
	void fillFloatArrayWithRandomNumbers(float* , int);
	void vectorAdditionCPU(const float*, const float*,float*, int);
	void cleanup(void);
	
	// variable declarations
	int size = iNumberOfArrayElements * sizeof(float);
	
	hipError_t  result = hipSuccess;
	
	// code
	// host memory allocation
	hostInput1 = (float*) malloc(size);
	if(hostInput1 == NULL){
	
		printf("Host memory allocation is failed for hostInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	
	}
	
	
	hostInput2 = (float*) malloc(size);
	if(hostInput2 == NULL){
	
		printf("Host memory allocation is failed for hostInput2 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	
	}

	hostOutput = (float*) malloc(size);
	if(hostOutput == NULL){
	
		printf("Host memory allocation is failed for hostOutput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	
	}
	
	gold = (float*) malloc(size);
	if(gold == NULL){
	
		printf("Host memory allocation is failed for gold array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	
	}
	
	// filling values into host arrays
	fillFloatArrayWithRandomNumbers(hostInput1, iNumberOfArrayElements);
	fillFloatArrayWithRandomNumbers(hostInput2, iNumberOfArrayElements);
	
	// device memory allocation
	result = hipMalloc((void**) &deviceInput1, size);
	if(result != hipSuccess){
		printf("Device memory allocation is failed for deviceInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	
	
	result = hipMalloc((void**)&deviceInput2, size);
	if(result != hipSuccess){
		
		printf("Device memory allocation is failed for deviceInput2 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	

	result = hipMalloc((void**)&deviceOutput, size);
	if(result != hipSuccess){
		printf("Device memory allocation is failed for deviceOutput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	
	}
	
	// copy data from host arrays into device arrays
	// cudaError_t cudaMemcpy(void * dest, const void * src, size_t count, enum cudaMemcpyKind)
	// copies count number of bytes from the mem. area pointed to by src to the mem. area pointed to by dest, where 
	// kind is one of the cudaMemcpyHostToHost, cudaMemcpyHostToDevice, cudaMemcpyDeviceToHost or cudaMemcpyDeviceToDevice.
	result = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	if(result != hipSuccess){
		printf("Host to Device data copy is failed for deviceInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	if(result != hipSuccess){
		printf("Host to Device data copy is failed for deviceInput2 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// CUDA kernel configuration
	// Note - ceil() i.e. ceiling func. will return the next int number closest to fractional number float. (256 - minimum number of threads on GPU.)
	dim3 dimGrid = dim3((int) ceil((float) iNumberOfArrayElements / 256.0f ), 1, 1); // y = 1, z = 1.
	dim3 dimBlock = dim3(256, 1, 1);

	// CUDA kernel for vector addition
	vectorAdditionGPU<<<dimGrid, dimBlock>>> (deviceInput1, deviceInput2, deviceOutput, iNumberOfArrayElements);

	// copy data from device array into host array
	result = hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
	if(result != hipSuccess){
		printf("Device to Host data copy is failed for hostOutput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	vectorAdditionCPU(hostInput1, hostInput2, gold, iNumberOfArrayElements );

	// comparison
	const float epsilon = 0.000001f;
	int breakValue = 1 ;
	bool bAccuracy = true;
	for(int i = 0 ; i < iNumberOfArrayElements; i++){

		float val1 = gold[i];
		float val2 = hostOutput[i];
		if(fabs(val1 - val2) > epsilon){
			bAccuracy = false;
			breakValue = i;
			break;

		}
	}

char str[128];
if (bAccuracy == false) {

	sprintf(str, "Comparison of CPU and GPU Vector Addition is not within accuracy of 0.000001 at array index %d", breakValue);
}
else {

	sprintf(str, "Comparison of CPU and GPU Vector Addition is within accuracy of 0.000001.");
}

// output
printf("Array1 begins from 0th index %.6f to %dth index %.6f\n", hostInput1[0], iNumberOfArrayElements - 1, hostInput1[iNumberOfArrayElements - 1]);

printf("Array2 begins from 0th index %.6f to %dth index %.6f\n", hostInput2[0], iNumberOfArrayElements - 1, hostInput2[iNumberOfArrayElements - 1]);

printf("CUDA kernel Grid dimension = %d,%d,%d and Block dimension = %d,%d,%d\n", dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

printf("Output array begind from 0th index of %.6f to %dth index %.6f\n", hostOutput[0], iNumberOfArrayElements - 1, hostOutput[iNumberOfArrayElements - 1]);

printf("%s\n", str);

//clean up
cleanup();

return(0);

}

void fillFloatArrayWithRandomNumbers(float* arr, int len){

	// code
	const float fscale = 1.0f / (float) RAND_MAX;
	for(int i = 0; i < len; i++){
		arr[i] = fscale * rand();
	}
}

void vectorAdditionCPU(const float* arr1, const float* arr2,float* out, int len){

	for(int i = 0; i < len; i++){

		out[i] = arr1[i] + arr2[i];
	}
}

void cleanup(void){

	// code
	if (deviceOutput) {

		hipFree(deviceOutput);
		deviceOutput = NULL;

	}

	if (deviceInput2) {

		hipFree(deviceInput2);
		deviceInput2 = NULL;
	}

	if (deviceInput1) {

		hipFree(deviceInput1);
		deviceInput1 = NULL;
	}

	if (gold) {

		hipFree(gold);
		gold = NULL;
	}

	if (hostOutput) {
		
		free(hostOutput);
		hostOutput = NULL;

	}

	if (hostInput2) {

		free(hostInput2);
		hostInput2 = NULL;

	}

	if (hostInput1) {

		free(hostInput1);
		hostInput1 = NULL;
	}

}